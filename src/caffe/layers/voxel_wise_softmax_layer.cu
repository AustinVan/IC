#include "hip/hip_runtime.h"
/*
 *
 *  Copyright (c) 2016, Facebook, Inc. All rights reserved.
 *
 *  Licensed under the Creative Commons Attribution-NonCommercial 3.0
 *  License (the "License"). You may obtain a copy of the License at
 *  https://creativecommons.org/licenses/by-nc/3.0/.
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  License for the specific language governing permissions and limitations
 *  under the License.
 *
 */



#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/voxel_wise_softmax_layer.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {



template <typename Dtype>
Dtype VoxelWiseSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  return Forward_cpu(bottom, top);
}

template <typename Dtype>
void VoxelWiseSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(VoxelWiseSoftmaxLayer);


}  // namespace caffe
